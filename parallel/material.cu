#include "hip/hip_runtime.h"
#include <math.h>					// sqrtf, powf
#include <hiprand/hiprand_kernel.h>			// hiprandState, hiprand_init
#include "header.h"

__device__ bool lambert_scatter(ray r, hit_record rec, vec3* attenuation, ray* scattered) {
	vec3 target = add(rec.p, add(rec.n, random_in_unit_sphere()));
	ray temp = {rec.p, sub(target, rec.p)};
	*scattered = temp;
	*attenuation = rec.s->albedo;
	return true;
}

__device__ vec3 reflect(vec3 v, vec3 n) {
	// v - 2*dot(v,n)*n
	return sub(v, smul(n, 2*dot(v, n)));
}

__device__ bool metal_scatter(ray r, hit_record rec, vec3* attenuation, ray* scattered) {
	vec3 reflected = reflect(unit(get_direction(r)), rec.n);
	// ray temp = {rec.p, reflected};
	ray temp = {rec.p, add(reflected, smul(random_in_unit_sphere(), rec.s->fuzz))};
	*scattered = temp;
	*attenuation = rec.s->albedo;
	return dot(get_direction(*scattered), rec.n) > 0;
}

__device__ bool refract(vec3 v, vec3 n, float ni_over_nt, vec3* refracted) {
	vec3 uv = unit(v);
	float dt = dot(uv, n);
	float discriminant = 1.0 - ni_over_nt*ni_over_nt * (1.0 - dt*dt);

	if (discriminant > 0.0) {
		// refracted = ni_over_nt * (uv - n*dt) - n*sqrtf(discriminant)
		*refracted = sub(smul(sub(uv, smul(n, dt)), ni_over_nt), smul(n, sqrtf(discriminant)));
		return true;
	}

	return false;
}

__device__ float schlick(float cosine, float refraction) {
	float r0 = (1.0 - refraction) / (1.0 + refraction);
	r0 = r0*r0;
	return r0 + (1.0 - r0) * powf(1.0 - cosine, 5.0);
}

__device__ bool glass_scatter(ray r, hit_record rec, vec3* attenuation, ray* scattered) {
	vec3 outward;
	vec3 reflected = reflect(get_direction(r), rec.n);
	float ni_over_nt;
	vec3 white = {1.0, 1.0, 1.0};
	*attenuation = white;
	vec3 refracted;

	float reflect_prob;
	float cosine;

	if (dot(get_direction(r), rec.n) > 0.0) {
		outward = smul(rec.n, -1.0);
		ni_over_nt = rec.s->refraction;

		cosine = rec.s->refraction * dot(get_direction(r), rec.n) / len(get_direction(r));
	} else {
		outward = rec.n;
		ni_over_nt = 1.0 / rec.s->refraction;

		cosine = (-1.0) * dot(get_direction(r), rec.n) / len(get_direction(r));
	}

	if (refract(get_direction(r), outward, ni_over_nt, &refracted)) {
		reflect_prob = schlick(cosine, rec.s->refraction);
	} else {
		ray temp = {rec.p, reflected};
		*scattered = temp;
		reflect_prob = 1.0;
	}

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int idx = j*nx + i;

	hiprandState state;
	hiprand_init(0, count[idx]++, 0, &state);

	if (GPU_RAND_FLT < reflect_prob) {
		ray temp = {rec.p, reflected};
		*scattered = temp;
	} else {
		ray temp = {rec.p, refracted};
		*scattered = temp;
	}

	return true;
}